#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


#define N 10000                   // Size of the grid
#define numSteps 1000           // Number of iteration
#define ALPHA 0.1               // Heat equation constant
#define SQUARE_SIZE 1000          // Dimension of the initial heated square
#define BLOCK_SIZE 32              // Thread block size for kernel
#define OutputNum 200           // Number of iterations to print grid

__global__ void heatEquation(float* u_old, float* u_new, const float dx2, const float dy2, const float dt) {
    // Thread indexes in the grid
    int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int j = blockIdx.y * blockDim.y + threadIdx.y + 1;

    if (i < N - 1 && j < N - 1) {
        int idx = j * N + i;     // Linear index

        // Heat equation
        u_new[idx] = u_old[idx] + ALPHA * dt * ( (u_old[idx - 1] - 2.0 * u_old[idx] + u_old[idx + 1])/dx2 + 
                                                 (u_old[idx - N] - 2.0 * u_old[idx] + u_old[idx + N])/dy2 );
    }
}

void GridToFile(const char* filename, float* grid) {
    FILE* file = fopen(filename, "w");
    if (file == NULL) {
        printf("Failed to open file '%s' for writing.\n", filename);
        return;
    }

    for (int j = 0; j < N; j++) {
        for (int i = 0; i < N; i++) {

            fprintf(file, "%f ", grid[j * N + i]);
        }

        fprintf(file, "\n");
    }

    fclose(file);
}

int main() {

    // Deltas
    const float dx = 0.01;   // Horizontal grid spacing 
    const float dy = 0.01;   // Vertical grid spacing

    const float dx2 = dx*dx;
    const float dy2 = dy*dy;

    const float dt = dx2 * dy2 / (2.0 * ALPHA * (dx2 + dy2)); // Largest stable time step

    // Declaring the inital and final grids
    float* u_old, *u_new;
    float* d_u_old, *d_u_new;

    u_old = (float*)malloc(N * N * sizeof(float));
    u_new = (float*)malloc(N * N * sizeof(float));

    // Initialize the grid
    for (int j = 0; j < N; j++) {
        for (int i = 0; i < N; i++) {
            int idx = j * N + i;
            if (i >= (N - SQUARE_SIZE) / 2 && i < (N + SQUARE_SIZE) / 2 &&
                j >= (N - SQUARE_SIZE) / 2 && j < (N + SQUARE_SIZE) / 2) {
                u_old[idx] = 1.0;
            } else {
                u_old[idx] = 0.0;
            }
        }
    }

    // // Writing the initial grid to a file
    // GridToFile("heat_0000.txt", u_old);

    // Allocating memory on the GPU
    hipMalloc((void**)&d_u_old, N * N * sizeof(float));
    hipMalloc((void**)&d_u_new, N * N * sizeof(float));

    // Copying the initial grid to the GPU
    hipMemcpy(d_u_old, u_old, N * N * sizeof(float), hipMemcpyHostToDevice);

    // Threaed block size and thread block's grid size
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((N-1) / (dimBlock.x + 1), (N-1) / (dimBlock.y + 1));

    // Declaring the variables to measure execssution time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Starting the timer
    hipEventRecord(start);

    // Launching the kernel
    for (int t = 0; t <= numSteps; t++) {
        heatEquation<<<dimGrid, dimBlock>>>(d_u_old, d_u_new, dx2, dy2, dt);

        // Swap the old and new grids
        float* temp = d_u_new;
        d_u_new = d_u_old;
        d_u_old = temp;

        // // Output visualization
        // if (t % OutputNum == 0 && t != 0){
        //     // Copying the final grid back to the CPU
        //     cudaMemcpy(u_new, d_u_old, N * N * sizeof(float), cudaMemcpyDeviceToHost);
        //     // Writing the final grid to a file
        //     char filename[64];
        //     sprintf(filename, "heat_%04d.txt", t);
        //     GridToFile(filename, u_new);
        // }
    }

    // Stopping the timer
    hipEventRecord(stop);

    // Synchronizing threads
    hipEventSynchronize(stop);

    // Saving the execution time
    float exec_time = 0;
    hipEventElapsedTime(&exec_time, start, stop);
    exec_time /= 1000;  //Execution time in seconds

    // Copying the final grid back to the CPU
    hipMemcpy(u_new, d_u_old, N * N * sizeof(float), hipMemcpyDeviceToHost);

    // Showing the parallel execution time
    printf("Parallel execution time: %f s \n", exec_time);

    // Free memory
    free(u_old);
    free(u_new);
    hipFree(d_u_old);
    hipFree(d_u_new);

    return 0;
}

